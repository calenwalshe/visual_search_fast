//
//  Academic License - for use in teaching, academic research, and meeting
//  course requirements at degree granting institutions only.  Not for
//  government, commercial, or other organizational use.
//
//  covert_search_dp_gpu_terminate.cu
//
//  Code generation for function 'covert_search_dp_gpu_terminate'
//


// Include files
#include "covert_search_dp_gpu_terminate.h"
#include "_coder_covert_search_dp_gpu_mex.h"
#include "covert_search_dp_gpu_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void covert_search_dp_gpu_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void covert_search_dp_gpu_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(errCode, hipGetErrorName(errCode), hipGetErrorString
                       (errCode), "SafeBuild", emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (covert_search_dp_gpu_terminate.cu)
