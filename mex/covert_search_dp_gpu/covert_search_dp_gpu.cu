#include "hip/hip_runtime.h"
//
//  Academic License - for use in teaching, academic research, and meeting
//  course requirements at degree granting institutions only.  Not for
//  government, commercial, or other organizational use.
//
//  covert_search_dp_gpu.cu
//
//  Code generation for function 'covert_search_dp_gpu'
//


// Include files
#include "covert_search_dp_gpu.h"
#include "covert_search_dp_gpu_data.h"
#include "covert_search_dp_gpu_emxutil.h"
#include "covert_search_dp_gpu_mexutil.h"
#include "covert_search_dp_gpu_types.h"
#include "rt_nonfinite.h"
#include "tic.h"
#include "toc.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "rt_nonfinite.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <math.h>
#include <stddef.h>
#include <string.h>

// Variable Definitions
static emlrtMCInfo emlrtMCI = { 34,    // lineNo
  9,                                   // colNo
  "rng",                               // fName
  "/usr/local/MATLAB/R2020b/toolbox/eml/lib/matlab/randfun/rng.m"// pName
};

static emlrtMCInfo b_emlrtMCI = { 99,  // lineNo
  17,                                  // colNo
  "fileopen",                          // fName
  "/usr/local/MATLAB/R2020b/toolbox/eml/lib/matlab/iofun/private/fileManager.m"// pName 
};

static emlrtMCInfo c_emlrtMCI = { 350, // lineNo
  1,                                   // colNo
  "freadExtrinsic",                    // fName
  "/usr/local/MATLAB/R2020b/toolbox/eml/lib/matlab/iofun/fread.m"// pName
};

static emlrtMCInfo d_emlrtMCI = { 354, // lineNo
  9,                                   // colNo
  "freadExtrinsic",                    // fName
  "/usr/local/MATLAB/R2020b/toolbox/eml/lib/matlab/iofun/fread.m"// pName
};

static emlrtMCInfo e_emlrtMCI = { 378, // lineNo
  13,                                  // colNo
  "freadExtrinsic",                    // fName
  "/usr/local/MATLAB/R2020b/toolbox/eml/lib/matlab/iofun/fread.m"// pName
};

// Function Declarations
static __device__ real_T atomicOpreal_T(real_T *address, real_T value);
static void b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId, real_T y_data[], int32_T y_size[2]);
static void b_emlrt_marshallIn(const mxArray *t, const char_T *identifier,
  real_T y_data[], int32_T y_size[2]);
static void c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId, real_T ret_data[], int32_T ret_size[2]);
static void checkCudaError(hipError_t errCode, const char_T *file, uint32_T
  line);
static __global__ void covert_search_dp_gpu_kernel1(const real_T dpmap_data
  [5760000], const int32_T dpmap_size[2], real_T priorh_data[5760000]);
static __global__ void covert_search_dp_gpu_kernel10(const emxArray_int32_T idx,
  const int32_T A_size[2], emxArray_real_T dv1);
static __global__ void covert_search_dp_gpu_kernel11(const emxArray_int8_T
  tpaMat, const emxArray_real_T dv1, const int32_T dv1_dim1, emxArray_int8_T
  b_tpaMat);
static __global__ void covert_search_dp_gpu_kernel12(const emxArray_int8_T
  tpaMat, const int32_T tpaMat_dim0, emxArray_int8_T b_tpaMat);
static __global__ void covert_search_dp_gpu_kernel13(const real_T priorh_data
  [5760000], real_T *nrows);
static __global__ void covert_search_dp_gpu_kernel14(const real_T priorh_data
  [5760000], int32_T pEnd, real_T *nrows);
static __global__ void covert_search_dp_gpu_kernel15(const uint32_T dv2[2],
  emxArray_real_T idx);
static __global__ void covert_search_dp_gpu_kernel16(const int32_T priorh_size[2],
  const int32_T pEnd, emxArray_real_T idx);
static __global__ void covert_search_dp_gpu_kernel17(const int32_T priorh_size[2],
  int16_T sizeA[2]);
static __global__ void covert_search_dp_gpu_kernel18(const emxArray_real_T idx,
  const int32_T idx_dim0, emxArray_int32_T v1);
static __global__ void covert_search_dp_gpu_kernel19(const int32_T p, const
  emxArray_int32_T v1, const int32_T v1_dim0, emxArray_int32_T vk);
static __global__ void covert_search_dp_gpu_kernel2(const int32_T dpmap_size[2],
  int16_T sizeA[2]);
static __global__ void covert_search_dp_gpu_kernel20(const int32_T p, const
  emxArray_int32_T vk, const int32_T v1_dim0, emxArray_int32_T v1);
static __global__ void covert_search_dp_gpu_kernel21(const emxArray_int32_T v1,
  const int32_T v1_dim0, emxArray_int32_T pre_yt);
static __global__ void covert_search_dp_gpu_kernel22(const emxArray_int32_T vk,
  const int32_T vk_dim0, emxArray_int32_T pre_xt);
static __global__ void covert_search_dp_gpu_kernel3(const real_T priorh_data
  [5760000], const int32_T A_size[2], const int32_T dpmap_size[2], const int16_T
  sizeA[2], const int32_T pEnd, real_T dpmap_data[5760000]);
static __global__ void covert_search_dp_gpu_kernel4(const int32_T pEnd,
  emxArray_int8_T tpaMat);
static __global__ void covert_search_dp_gpu_kernel5(const real_T *nrows, const
  int32_T pEnd, emxArray_int8_T tpaMat);
static __global__ void covert_search_dp_gpu_kernel6(const int32_T dv1_dim1,
  emxArray_int32_T idx);
static __global__ void covert_search_dp_gpu_kernel7(const emxArray_real_T dv1,
  const int32_T pEnd, emxArray_int32_T idx);
static __global__ void covert_search_dp_gpu_kernel8(const int32_T dv1_dim1,
  emxArray_int32_T idx);
static __global__ void covert_search_dp_gpu_kernel9(const emxArray_int32_T iwork,
  const int32_T j, const int32_T kEnd, emxArray_int32_T idx);
static void d_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId, real_T ret_data[], int32_T ret_size[2]);
static __device__ int32_T div_s32_device(int32_T numerator, int32_T denominator);
static void emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId, real_T y_data[], int32_T y_size[2]);
static void emlrt_marshallIn(const mxArray *t, const char_T *identifier, real_T
  y_data[], int32_T y_size[2]);
static const mxArray *feval(const mxArray *b, const mxArray *c, const mxArray *d,
  emlrtMCInfo *location);
static void feval(const mxArray *b, const mxArray *c, const mxArray *d, const
                  mxArray *e, const mxArray *f, emlrtMCInfo *location, const
                  mxArray **g, const mxArray **h);
static void gpuEmxFree_int32_T(emxArray_int32_T *gpu);
static void gpuEmxFree_int8_T(emxArray_int8_T *gpu);
static void gpuEmxFree_real_T(emxArray_real_T *gpu);
static void gpuEmxMemcpyCpuToGpu_int32_T(const emxArray_int32_T *cpu,
  emxArray_int32_T *gpu);
static void gpuEmxMemcpyCpuToGpu_int8_T(const emxArray_int8_T *cpu,
  emxArray_int8_T *gpu);
static void gpuEmxMemcpyCpuToGpu_real_T(const emxArray_real_T *cpu,
  emxArray_real_T *gpu);
static void gpuEmxMemcpyGpuToCpu_int32_T(emxArray_int32_T *cpu, emxArray_int32_T
  *gpu);
static void gpuEmxMemcpyGpuToCpu_int8_T(emxArray_int8_T *cpu, emxArray_int8_T
  *gpu);
static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T
  *gpu);
static void gpuEmxReset_int32_T(emxArray_int32_T *gpu);
static void gpuEmxReset_int8_T(emxArray_int8_T *gpu);
static void gpuEmxReset_real_T(emxArray_real_T *gpu);
static void rng(const mxArray *b, emlrtMCInfo *location);
static __device__ real_T shflDown2(real_T in1, uint32_T offset, uint32_T mask);
static const mxArray *size(const mxArray *b, const mxArray *c, emlrtMCInfo
  *location);
static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
  uint32_T mask);
static __device__ real_T workGroupReduction(real_T val, uint32_T mask, uint32_T
  numActiveWarps);

// Function Definitions
static __device__ real_T atomicOpreal_T(real_T *address, real_T value)
{
  unsigned long long int old;
  unsigned long long int *address_as_up;
  address_as_up = (unsigned long long int *)address;
  old = *address_as_up;
  unsigned long long int assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_up, old, __double_as_longlong(value +
      __longlong_as_double(old)));
  } while (assumed != old);

  return __longlong_as_double(old);
}

static void b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId, real_T y_data[], int32_T y_size[2])
{
  d_emlrt_marshallIn(emlrtAlias(u), parentId, y_data, y_size);
  emlrtDestroyArray(&u);
}

static void b_emlrt_marshallIn(const mxArray *t, const char_T *identifier,
  real_T y_data[], int32_T y_size[2])
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  b_emlrt_marshallIn(emlrtAlias(t), &thisId, y_data, y_size);
  emlrtDestroyArray(&t);
}

static void c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId, real_T ret_data[], int32_T ret_size[2])
{
  static const int32_T dims[2] = { 2400, 2400 };

  int32_T iv[2];
  const boolean_T bv[2] = { true, true };

  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 2U,
    dims, &bv[0], iv);
  ret_size[0] = iv[0];
  ret_size[1] = iv[1];
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, (void *)ret_data, 8, false);
  emlrtDestroyArray(&src);
}

static void checkCudaError(hipError_t errCode, const char_T *file, uint32_T
  line)
{
  emlrtRTEInfo rtInfo;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  if (errCode != hipSuccess) {
    uint32_T len;
    hipGetLastError();
    len = strlen(file);
    pn = (char_T *)calloc(len + 1U, 1U);
    fn = (char_T *)calloc(len + 1U, 1U);
    memcpy(pn, file, len);
    memcpy(fn, file, len);
    brk = strrchr(fn, '.');
    *brk = '\x00';
    brk = strrchr(fn, '/');
    if (brk == NULL) {
      brk = strrchr(fn, '\\');
    }

    if (brk == NULL) {
      brk = fn;
    } else {
      brk++;
    }

    rtInfo.lineNo = static_cast<int32_T>(line);
    rtInfo.colNo = 0;
    rtInfo.fName = brk;
    rtInfo.pName = pn;
    emlrtCUDAError(errCode, hipGetErrorName(errCode), hipGetErrorString
                   (errCode), &rtInfo, emlrtRootTLSGlobal);
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel1(
  const real_T dpmap_data[5760000], const int32_T dpmap_size[2], real_T
  priorh_data[5760000])
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(dpmap_size[0] * dpmap_size[1] - 1);
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    priorh_data[i] = dpmap_data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel10(
  const emxArray_int32_T idx, const int32_T A_size[2], emxArray_real_T dv1)
{
  int64_T loopEnd;
  uint64_T b_idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(A_size[0] * A_size[1] - 1);
  for (b_idx = threadId; b_idx <= static_cast<uint64_T>(loopEnd); b_idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(b_idx);
    dv1.data[i] = static_cast<real_T>(idx.data[i]);
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel11(
  const emxArray_int8_T tpaMat, const emxArray_real_T dv1, const int32_T
  dv1_dim1, emxArray_int8_T b_tpaMat)
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(dv1_dim1 - 1);
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    b_tpaMat.data[i] = tpaMat.data[static_cast<int32_T>(dv1.data[i]) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel12(
  const emxArray_int8_T tpaMat, const int32_T tpaMat_dim0, emxArray_int8_T
  b_tpaMat)
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(tpaMat_dim0 - 1);
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    b_tpaMat.data[i] = tpaMat.data[i];
  }
}

static __global__ __launch_bounds__(32, 1) void covert_search_dp_gpu_kernel13(
  const real_T priorh_data[5760000], real_T *nrows)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *nrows = priorh_data[0];
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel14(
  const real_T priorh_data[5760000], int32_T pEnd, real_T *nrows)
{
  int64_T loopEnd;
  real_T tmpRed0;
  uint32_T blockStride;
  uint32_T idx;
  uint32_T m;
  uint32_T thBlkId;
  uint32_T threadId;
  uint32_T threadStride;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  loopEnd = static_cast<int64_T>(pEnd - 2);
  if (mwIsLastBlock()) {
    m = (static_cast<int64_T>(pEnd - 2) + 1L) % static_cast<int64_T>(blockStride);
    if (m > 0U) {
      blockStride = m;
    }
  }

  blockStride = (uint32_T)(blockStride + (static_cast<int64_T>(warpSize) - 1L)) /
    warpSize;
  if (static_cast<int64_T>(threadId) <= loopEnd) {
    tmpRed0 = priorh_data[static_cast<int32_T>(threadId) + 1];
  }

  m = __ballot_sync(MAX_uint32_T, static_cast<int64_T>(threadId) <= loopEnd);
  for (idx = threadId + threadStride; idx <= static_cast<uint32_T>(loopEnd); idx
       += threadStride) {
    tmpRed0 += priorh_data[static_cast<int32_T>(idx) + 1];
  }

  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if (thBlkId == 0U) {
    atomicOpreal_T(&nrows[0], tmpRed0);
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel15(
  const uint32_T dv2[2], emxArray_real_T idx)
{
  int64_T loopEnd;
  uint64_T b_idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(static_cast<int32_T>(dv2[0]) - 1);
  for (b_idx = threadId; b_idx <= static_cast<uint64_T>(loopEnd); b_idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(b_idx);
    idx.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel16(
  const int32_T priorh_size[2], const int32_T pEnd, emxArray_real_T idx)
{
  int64_T loopEnd;
  uint64_T b_idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(pEnd - 1);
  for (b_idx = threadId; b_idx <= static_cast<uint64_T>(loopEnd); b_idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(b_idx);
    idx.data[i] = floor(idx.data[i] * static_cast<real_T>(priorh_size[0] *
      priorh_size[1])) + 1.0;
  }
}

static __global__ __launch_bounds__(32, 1) void covert_search_dp_gpu_kernel17(
  const int32_T priorh_size[2], int16_T sizeA[2])
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 2) {
    sizeA[i] = static_cast<int16_T>(priorh_size[i]);
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel18(
  const emxArray_real_T idx, const int32_T idx_dim0, emxArray_int32_T v1)
{
  int64_T loopEnd;
  uint64_T b_idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(idx_dim0 - 1);
  for (b_idx = threadId; b_idx <= static_cast<uint64_T>(loopEnd); b_idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(b_idx);
    v1.data[i] = static_cast<int32_T>(idx.data[i]) - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel19(
  const int32_T p, const emxArray_int32_T v1, const int32_T v1_dim0,
  emxArray_int32_T vk)
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(v1_dim0 - 1);
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    vk.data[i] = div_s32_device(v1.data[i], p);
  }
}

static __global__ __launch_bounds__(32, 1) void covert_search_dp_gpu_kernel2(
  const int32_T dpmap_size[2], int16_T sizeA[2])
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 2) {
    sizeA[i] = static_cast<int16_T>(dpmap_size[i]);
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel20(
  const int32_T p, const emxArray_int32_T vk, const int32_T v1_dim0,
  emxArray_int32_T v1)
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(v1_dim0 - 1);
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    v1.data[i] -= vk.data[i] * p;
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel21(
  const emxArray_int32_T v1, const int32_T v1_dim0, emxArray_int32_T pre_yt)
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(v1_dim0 - 1);
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    pre_yt.data[i] = v1.data[i] + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel22(
  const emxArray_int32_T vk, const int32_T vk_dim0, emxArray_int32_T pre_xt)
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(vk_dim0 - 1);
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    pre_xt.data[i] = vk.data[i] + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel3(
  const real_T priorh_data[5760000], const int32_T A_size[2], const int32_T
  dpmap_size[2], const int16_T sizeA[2], const int32_T pEnd, real_T dpmap_data
  [5760000])
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<int64_T>(static_cast<int32_T>(sizeA[0]) - 1) + 1L) * (
    static_cast<int64_T>(pEnd - 1) + 1L) - 1L;
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(static_cast<int32_T>
      (sizeA[0]) - 1) + 1UL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
      (static_cast<uint64_T>(static_cast<int32_T>(sizeA[0]) - 1) + 1UL));
    dpmap_data[i + dpmap_size[0] * j] = priorh_data[((static_cast<int32_T>
      (sizeA[0]) - i) + A_size[0] * ((static_cast<int32_T>(sizeA[1]) - j) - 1))
      - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel4(
  const int32_T pEnd, emxArray_int8_T tpaMat)
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(pEnd);
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    tpaMat.data[i] = static_cast<int8_T>(0);
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel5(
  const real_T *nrows, const int32_T pEnd, emxArray_int8_T tpaMat)
{
  int64_T loopEnd;
  uint64_T idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(pEnd);
  for (idx = threadId; idx <= static_cast<uint64_T>(loopEnd); idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    tpaMat.data[i + static_cast<int32_T>(*nrows)] = static_cast<int8_T>(1);
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel6(
  const int32_T dv1_dim1, emxArray_int32_T idx)
{
  int64_T loopEnd;
  uint64_T b_idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(dv1_dim1 - 1);
  for (b_idx = threadId; b_idx <= static_cast<uint64_T>(loopEnd); b_idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(b_idx);
    idx.data[i] = 0;
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel7(
  const emxArray_real_T dv1, const int32_T pEnd, emxArray_int32_T idx)
{
  int64_T loopEnd;
  uint64_T b_idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>((pEnd - 1) / 2);
  for (b_idx = threadId; b_idx <= static_cast<uint64_T>(loopEnd); b_idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(b_idx);
    i = (i << 1) + 1;
    if ((static_cast<int32_T>(dv1.data[i - 1] <= dv1.data[i])) ||
        (static_cast<int32_T>(isnan(dv1.data[i])))) {
      idx.data[i - 1] = i;
      idx.data[i] = i + 1;
    } else {
      idx.data[i - 1] = i + 1;
      idx.data[i] = i;
    }
  }
}

static __global__ __launch_bounds__(32, 1) void covert_search_dp_gpu_kernel8(
  const int32_T dv1_dim1, emxArray_int32_T idx)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    idx.data[dv1_dim1 - 1] = dv1_dim1;
  }
}

static __global__ __launch_bounds__(1024, 1) void covert_search_dp_gpu_kernel9(
  const emxArray_int32_T iwork, const int32_T j, const int32_T kEnd,
  emxArray_int32_T idx)
{
  int64_T loopEnd;
  uint64_T b_idx;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = mwGetGlobalThreadIndex();
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<int64_T>(kEnd - 1);
  for (b_idx = threadId; b_idx <= static_cast<uint64_T>(loopEnd); b_idx +=
       threadStride) {
    int32_T i;
    i = static_cast<int32_T>(b_idx);
    idx.data[(j + i) - 1] = iwork.data[i];
  }
}

static void d_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId, real_T ret_data[], int32_T ret_size[2])
{
  static const int32_T dims[2] = { 135, 135 };

  int32_T iv[2];
  const boolean_T bv[2] = { true, true };

  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 2U,
    dims, &bv[0], iv);
  ret_size[0] = iv[0];
  ret_size[1] = iv[1];
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, (void *)ret_data, 8, false);
  emlrtDestroyArray(&src);
}

static __device__ int32_T div_s32_device(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  uint32_T b_numerator;
  if (denominator == 0) {
    if (numerator >= 0) {
      quotient = MAX_int32_T;
    } else {
      quotient = MIN_int32_T;
    }
  } else {
    uint32_T b_denominator;
    if (numerator < 0) {
      b_numerator = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      b_numerator = static_cast<uint32_T>(numerator);
    }

    if (denominator < 0) {
      b_denominator = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b_denominator = static_cast<uint32_T>(denominator);
    }

    b_numerator /= b_denominator;
    if (static_cast<int32_T>(numerator < 0) != static_cast<int32_T>(denominator <
         0)) {
      quotient = -static_cast<int32_T>(b_numerator);
    } else {
      quotient = static_cast<int32_T>(b_numerator);
    }
  }

  return quotient;
}

static void emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId, real_T y_data[], int32_T y_size[2])
{
  c_emlrt_marshallIn(emlrtAlias(u), parentId, y_data, y_size);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *t, const char_T *identifier, real_T
  y_data[], int32_T y_size[2])
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(t), &thisId, y_data, y_size);
  emlrtDestroyArray(&t);
}

static void feval(const mxArray *b, const mxArray *c, const mxArray *d, const
                  mxArray *e, const mxArray *f, emlrtMCInfo *location, const
                  mxArray **g, const mxArray **h)
{
  const mxArray *pArrays[5];
  const mxArray *mv[2];
  pArrays[0] = b;
  pArrays[1] = c;
  pArrays[2] = d;
  pArrays[3] = e;
  pArrays[4] = f;
  emlrtAssign(g, emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 2, &mv[0], 5, pArrays,
    "feval", true, location));
  emlrtAssign(h, mv[1]);
}

static const mxArray *feval(const mxArray *b, const mxArray *c, const mxArray *d,
  emlrtMCInfo *location)
{
  const mxArray *pArrays[3];
  const mxArray *m;
  pArrays[0] = b;
  pArrays[1] = c;
  pArrays[2] = d;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 3, pArrays, "feval",
    true, location);
}

static void gpuEmxFree_int32_T(emxArray_int32_T *gpu)
{

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->data != (void *)4207599121UL) {
    CUDACHECK(hipFree(gpu->data));
  }

  emlrtFreeMex(gpu->size);

#undef CUDACHECK

}

static void gpuEmxFree_int8_T(emxArray_int8_T *gpu)
{

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->data != (void *)4207599121UL) {
    CUDACHECK(hipFree(gpu->data));
  }

  emlrtFreeMex(gpu->size);

#undef CUDACHECK

}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->data != (void *)4207599121UL) {
    CUDACHECK(hipFree(gpu->data));
  }

  emlrtFreeMex(gpu->size);

#undef CUDACHECK

}

static void gpuEmxMemcpyCpuToGpu_int32_T(const emxArray_int32_T *cpu,
  emxArray_int32_T *gpu)
{
  int32_T actualSize;
  int32_T i;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = (int32_T *)emlrtCallocMex(gpu->numDimensions, sizeof(int32_T));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }

  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      CUDACHECK(hipFree(gpu->data));
    }

    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }

    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    CUDACHECK(hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(int32_T)));
  }

  CUDACHECK(hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(int32_T),
                       hipMemcpyHostToDevice));

#undef CUDACHECK

}

static void gpuEmxMemcpyCpuToGpu_int8_T(const emxArray_int8_T *cpu,
  emxArray_int8_T *gpu)
{
  int32_T actualSize;
  int32_T i;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = (int32_T *)emlrtCallocMex(gpu->numDimensions, sizeof(int32_T));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }

  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      CUDACHECK(hipFree(gpu->data));
    }

    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }

    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    CUDACHECK(hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(int8_T)));
  }

  CUDACHECK(hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(int8_T),
                       hipMemcpyHostToDevice));

#undef CUDACHECK

}

static void gpuEmxMemcpyCpuToGpu_real_T(const emxArray_real_T *cpu,
  emxArray_real_T *gpu)
{
  int32_T actualSize;
  int32_T i;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = (int32_T *)emlrtCallocMex(gpu->numDimensions, sizeof(int32_T));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }

  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      CUDACHECK(hipFree(gpu->data));
    }

    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }

    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    CUDACHECK(hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(real_T)));
  }

  CUDACHECK(hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(real_T),
                       hipMemcpyHostToDevice));

#undef CUDACHECK

}

static void gpuEmxMemcpyGpuToCpu_int32_T(emxArray_int32_T *cpu, emxArray_int32_T
  *gpu)
{
  int32_T actualSize;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  actualSize = 1;
  for (int32_T i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }

  CUDACHECK(hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(int32_T),
                       hipMemcpyDeviceToHost));

#undef CUDACHECK

}

static void gpuEmxMemcpyGpuToCpu_int8_T(emxArray_int8_T *cpu, emxArray_int8_T
  *gpu)
{
  int32_T actualSize;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  actualSize = 1;
  for (int32_T i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }

  CUDACHECK(hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(int8_T),
                       hipMemcpyDeviceToHost));

#undef CUDACHECK

}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T
  *gpu)
{
  int32_T actualSize;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  actualSize = 1;
  for (int32_T i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }

  CUDACHECK(hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(real_T),
                       hipMemcpyDeviceToHost));

#undef CUDACHECK

}

static void gpuEmxReset_int32_T(emxArray_int32_T *gpu)
{
  memset(gpu, 0, sizeof(emxArray_int32_T));
}

static void gpuEmxReset_int8_T(emxArray_int8_T *gpu)
{
  memset(gpu, 0, sizeof(emxArray_int8_T));
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  memset(gpu, 0, sizeof(emxArray_real_T));
}

static void rng(const mxArray *b, emlrtMCInfo *location)
{
  const mxArray *pArray;
  pArray = b;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 1, &pArray, "rng", true,
                        location);
}

static __device__ real_T shflDown2(real_T in1, uint32_T offset, uint32_T mask)
{
  int2 tmp;
  tmp = *(int2 *)&in1;
  tmp.x = __shfl_down_sync(mask, tmp.x, offset);
  tmp.y = __shfl_down_sync(mask, tmp.y, offset);
  return *(real_T *)&tmp;
}

static const mxArray *size(const mxArray *b, const mxArray *c, emlrtMCInfo
  *location)
{
  const mxArray *pArrays[2];
  const mxArray *m;
  pArrays[0] = b;
  pArrays[1] = c;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 2, pArrays, "size",
    true, location);
}

static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
  uint32_T mask)
{
  uint32_T activeSize;
  uint32_T offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    real_T other;
    other = shflDown2(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }

    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }

  return val;
}

static __device__ real_T workGroupReduction(real_T val, uint32_T mask, uint32_T
  numActiveWarps)
{
  __shared__ real_T shared[32];
  uint32_T lane;
  uint32_T thBlkId;
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  lane = thBlkId % warpSize;
  thBlkId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[thBlkId] = val;
  }

  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (thBlkId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }

  return val;
}

void covert_search_dp_gpu(covert_search_dp_gpuStackData *SD, real_T trials,
  const char_T dpmapfp[9], const char_T priorhfp[10])
{
  static const int32_T iv[2] = { 1, 5 };

  static const int32_T iv1[2] = { 1, 9 };

  static const int32_T iv10[2] = { 1, 5 };

  static const int32_T iv11[2] = { 1, 8 };

  static const int32_T iv12[2] = { 1, 5 };

  static const int32_T iv13[2] = { 1, 2 };

  static const int32_T iv14[2] = { 1, 6 };

  static const int32_T iv2[2] = { 1, 5 };

  static const int32_T iv3[2] = { 1, 2 };

  static const int32_T iv4[2] = { 1, 6 };

  static const int32_T iv5[2] = { 1, 5 };

  static const int32_T iv6[2] = { 1, 10 };

  static const int32_T iv7[2] = { 1, 5 };

  static const int32_T iv8[2] = { 1, 2 };

  static const int32_T iv9[2] = { 1, 6 };

  static const char_T filename[8] = { 'r', 't', 'm', 'p', '.', 'b', 'i', 'n' };

  static const char_T precision[6] = { 'd', 'o', 'u', 'b', 'l', 'e' };

  static const char_T b_u[5] = { 'f', 'r', 'e', 'a', 'd' };

  static const char_T u[5] = { 'f', 'o', 'p', 'e', 'n' };

  dim3 b_block;
  dim3 b_grid;
  dim3 block;
  dim3 c_block;
  dim3 c_grid;
  dim3 d_block;
  dim3 d_grid;
  dim3 e_block;
  dim3 e_grid;
  dim3 f_block;
  dim3 f_grid;
  dim3 g_block;
  dim3 g_grid;
  dim3 grid;
  dim3 h_block;
  dim3 h_grid;
  dim3 i_block;
  dim3 i_grid;
  dim3 j_block;
  dim3 j_grid;
  dim3 k_block;
  dim3 k_grid;
  dim3 l_block;
  dim3 l_grid;
  dim3 m_block;
  dim3 m_grid;
  dim3 n_block;
  dim3 n_grid;
  dim3 o_block;
  dim3 o_grid;
  dim3 p_block;
  dim3 p_grid;
  dim3 q_block;
  dim3 q_grid;
  dim3 r_block;
  dim3 r_grid;
  emxArray_int32_T gpu_idx;
  emxArray_int32_T gpu_iwork;
  emxArray_int32_T gpu_pre_xt;
  emxArray_int32_T gpu_pre_yt;
  emxArray_int32_T gpu_v1;
  emxArray_int32_T gpu_vk;
  emxArray_int32_T *idx;
  emxArray_int32_T *iwork;
  emxArray_int32_T *pre_xt;
  emxArray_int32_T *pre_yt;
  emxArray_int32_T *v1;
  emxArray_int32_T *vk;
  emxArray_int8_T b_gpu_tpaMat;
  emxArray_int8_T gpu_tpaMat;
  emxArray_int8_T *b_tpaMat;
  emxArray_int8_T *tpaMat;
  emxArray_real_T b_gpu_idx;
  emxArray_real_T gpu_dv1;
  emxArray_real_T *b_idx;
  emxArray_real_T *dv1;
  emxArray_real_T *r;
  const mxArray *ab_y;
  const mxArray *b_count;
  const mxArray *b_t;
  const mxArray *b_y;
  const mxArray *bb_y;
  const mxArray *c_count;
  const mxArray *c_t;
  const mxArray *c_y;
  const mxArray *cb_y;
  const mxArray *count;
  const mxArray *d_y;
  const mxArray *db_y;
  const mxArray *e_y;
  const mxArray *eb_y;
  const mxArray *f_y;
  const mxArray *fb_y;
  const mxArray *g_y;
  const mxArray *h_y;
  const mxArray *i_y;
  const mxArray *j_y;
  const mxArray *k_y;
  const mxArray *l_y;
  const mxArray *m;
  const mxArray *m_y;
  const mxArray *n_y;
  const mxArray *o_y;
  const mxArray *p_y;
  const mxArray *q_y;
  const mxArray *r_y;
  const mxArray *s_y;
  const mxArray *t;
  const mxArray *t_y;
  const mxArray *u_y;
  const mxArray *v_y;
  const mxArray *w_y;
  const mxArray *x_y;
  const mxArray *y;
  const mxArray *y_y;
  real_T (*gpu_dpmap_data)[5760000];
  real_T (*gpu_priorh_data)[5760000];
  real_T rtmp_data[18225];
  real_T (*gpu_rtmp_data)[18225];
  real_T ncols;
  real_T nrows;
  real_T *gpu_nrows;
  real_T *pData;
  int32_T A_size[2];
  int32_T dpmap_size[2];
  int32_T edges_size[2];
  int32_T priorh_size[2];
  int32_T rtmp_size[2];
  int32_T x_size[2];
  int32_T (*gpu_A_size)[2];
  int32_T (*gpu_dpmap_size)[2];
  int32_T (*gpu_priorh_size)[2];
  int32_T (*gpu_rtmp_size)[2];
  int32_T (*gpu_x_size)[2];
  int32_T i;
  int32_T i2;
  int32_T j;
  int32_T k;
  int32_T kEnd;
  int32_T n;
  int32_T p;
  int32_T pEnd;
  int32_T q;
  int32_T qEnd;
  int32_T rt;
  uint32_T dv2[2];
  uint32_T (*gpu_dv2)[2];
  int16_T sizeA[2];
  int16_T (*gpu_sizeA)[2];
  boolean_T A_size_dirtyOnCpu;
  boolean_T dpmap_data_dirtyOnCpu;
  boolean_T dpmap_size_dirtyOnCpu;
  boolean_T dv1_dirtyOnCpu;
  boolean_T idx_dirtyOnCpu;
  boolean_T idx_dirtyOnGpu;
  boolean_T iwork_dirtyOnCpu;
  boolean_T nrows_dirtyOnCpu;
  boolean_T priorh_data_dirtyOnCpu;
  boolean_T priorh_data_dirtyOnGpu;
  boolean_T priorh_size_dirtyOnCpu;
  boolean_T rtmp_data_dirtyOnCpu;
  boolean_T rtmp_size_dirtyOnCpu;
  boolean_T sizeA_dirtyOnGpu;
  boolean_T tpaMat_dirtyOnCpu;
  boolean_T tpaMat_dirtyOnGpu;
  boolean_T validLaunchParams;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  CUDACHECK(hipMalloc(&gpu_rtmp_data, 145800UL));
  CUDACHECK(hipMalloc(&gpu_x_size, 8UL));
  gpuEmxReset_int32_T(&gpu_pre_xt);
  gpuEmxReset_int32_T(&gpu_pre_yt);
  gpuEmxReset_int32_T(&gpu_vk);
  gpuEmxReset_int32_T(&gpu_v1);
  CUDACHECK(hipMalloc(&gpu_dv2, 8UL));
  gpuEmxReset_real_T(&b_gpu_idx);
  gpuEmxReset_int8_T(&b_gpu_tpaMat);
  gpuEmxReset_int32_T(&gpu_iwork);
  gpuEmxReset_int32_T(&gpu_idx);
  gpuEmxReset_real_T(&gpu_dv1);
  gpuEmxReset_int8_T(&gpu_tpaMat);
  CUDACHECK(hipMalloc(&gpu_rtmp_size, 8UL));
  CUDACHECK(hipMalloc(&gpu_priorh_size, 8UL));
  CUDACHECK(hipMalloc(&gpu_priorh_data, 46080000UL));
  CUDACHECK(hipMalloc(&gpu_sizeA, 4UL));
  CUDACHECK(hipMalloc(&gpu_dpmap_data, 46080000UL));
  CUDACHECK(hipMalloc(&gpu_A_size, 8UL));
  CUDACHECK(hipMalloc(&gpu_dpmap_size, 8UL));
  CUDACHECK(hipMalloc(&gpu_nrows, 8UL));
  rtmp_data_dirtyOnCpu = false;
  priorh_data_dirtyOnCpu = false;
  dpmap_data_dirtyOnCpu = false;
  priorh_data_dirtyOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);

  // codegen
  //  COVERT_SEARCH_DP runs the search model with specified parameters and
  //  number of trials.
  //
  //  COVERT_SEARCH_DP calls the following routines:
  //  mk_prior   make the prior prob distribution of target location
  //  mk_amap    make a map of target amplitude (may change if contrast fixed)
  //  mk_fov     make the scalar foveation map
  //  mk_tarloc  randomly sample target location from prior
  //  mk_lr      compute log likelihood ratio
  //
  // note that (i,j) = (y,x)
  //  note:  xcorr2 = conv2(a, rot90(conj(b),2));  can add 'same' to this
  // coder.gpu.kernelfun;
  coder::tic();
  y = NULL;
  m = emlrtCreateDoubleScalar(1.0);
  emlrtAssign(&y, m);
  rng(y, &emlrtMCI);
  b_y = NULL;
  m = emlrtCreateCharArray(2, &iv[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m, &u[0]);
  emlrtAssign(&b_y, m);
  c_y = NULL;
  m = emlrtCreateCharArray(2, &iv1[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 9, m, &dpmapfp[0]);
  emlrtAssign(&c_y, m);
  d_y = NULL;
  m = emlrtCreateString1('r');
  emlrtAssign(&d_y, m);
  nrows = emlrt_marshallIn(feval(b_y, c_y, d_y, &b_emlrtMCI),
    "<output of feval>");
  t = NULL;
  count = NULL;
  e_y = NULL;
  m = emlrtCreateCharArray(2, &iv2[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m, &b_u[0]);
  emlrtAssign(&e_y, m);
  f_y = NULL;
  m = emlrtCreateDoubleScalar(nrows);
  emlrtAssign(&f_y, m);
  g_y = NULL;
  m = emlrtCreateNumericArray(2, &iv3[0], mxDOUBLE_CLASS, mxREAL);
  pData = emlrtMxGetPr(m);
  for (pEnd = 0; pEnd < 2; pEnd++) {
    pData[pEnd] = 2400.0;
  }

  emlrtAssign(&g_y, m);
  h_y = NULL;
  m = emlrtCreateCharArray(2, &iv4[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 6, m, &precision[0]);
  emlrtAssign(&h_y, m);
  i_y = NULL;
  m = emlrtCreateNumericMatrix(1, 1, mxINT32_CLASS, mxREAL);
  *(int32_T *)emlrtMxGetData(m) = 0;
  emlrtAssign(&i_y, m);
  feval(e_y, f_y, g_y, h_y, i_y, &c_emlrtMCI, &t, &count);
  emlrt_marshallIn(emlrtAlias(count), "count");
  j_y = NULL;
  m = emlrtCreateDoubleScalar(1.0);
  emlrtAssign(&j_y, m);
  nrows = emlrt_marshallIn(size(emlrtAlias(t), j_y, &d_emlrtMCI),
    "<output of size>");
  k_y = NULL;
  m = emlrtCreateDoubleScalar(2.0);
  emlrtAssign(&k_y, m);
  ncols = emlrt_marshallIn(size(emlrtAlias(t), k_y, &e_emlrtMCI),
    "<output of size>");
  if (nrows * ncols == 0.0) {
    dpmap_size[0] = 2400;
    dpmap_size[1] = 0;
    dpmap_size_dirtyOnCpu = true;
  } else {
    emlrt_marshallIn(emlrtAlias(t), "t", SD->f0.dpmap_data, dpmap_size);
    dpmap_size_dirtyOnCpu = true;
    dpmap_data_dirtyOnCpu = true;
  }

  emlrtDestroyArray(&t);
  emlrtDestroyArray(&count);
  A_size[0] = dpmap_size[0];
  A_size[1] = dpmap_size[1];
  A_size_dirtyOnCpu = true;
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((dpmap_size[0] *
    dpmap_size[1] - 1) + 1L), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    if (dpmap_data_dirtyOnCpu) {
      CUDACHECK(hipMemcpy(gpu_dpmap_data, &SD->f0.dpmap_data[0], 46080000UL,
                           hipMemcpyHostToDevice));
      dpmap_data_dirtyOnCpu = false;
    }

    CUDACHECK(hipMemcpy(gpu_dpmap_size, &dpmap_size[0], 8UL,
                         hipMemcpyHostToDevice));
    dpmap_size_dirtyOnCpu = false;
    covert_search_dp_gpu_kernel1<<<grid, block>>>(*gpu_dpmap_data,
      *gpu_dpmap_size, *gpu_priorh_data);
    CUDACHECK(hipGetLastError());
    priorh_data_dirtyOnGpu = true;
  }

  if ((dpmap_size[0] != 0) && (dpmap_size[1] != 0)) {
    if (dpmap_size_dirtyOnCpu) {
      CUDACHECK(hipMemcpy(gpu_dpmap_size, &dpmap_size[0], 8UL,
                           hipMemcpyHostToDevice));
    }

    covert_search_dp_gpu_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_dpmap_size, *gpu_sizeA);
    CUDACHECK(hipGetLastError());
    CUDACHECK(hipMemcpy(&sizeA[0], gpu_sizeA, 4UL, hipMemcpyDeviceToHost));
    dpmap_size[0] = sizeA[0];
    dpmap_size[1] = sizeA[1];
    dpmap_size_dirtyOnCpu = true;
    pEnd = sizeA[1];
    validLaunchParams = mwGetLaunchParameters(static_cast<real_T>(((sizeA[0] - 1)
      + 1L) * ((pEnd - 1) + 1L)), &b_grid, &b_block, 1024U, 65535U);
    if (validLaunchParams) {
      if (dpmap_data_dirtyOnCpu) {
        CUDACHECK(hipMemcpy(gpu_dpmap_data, &SD->f0.dpmap_data[0], 46080000UL,
                             hipMemcpyHostToDevice));
      }

      CUDACHECK(hipMemcpy(gpu_dpmap_size, &dpmap_size[0], 8UL,
                           hipMemcpyHostToDevice));
      dpmap_size_dirtyOnCpu = false;
      CUDACHECK(hipMemcpy(gpu_A_size, &A_size[0], 8UL, hipMemcpyHostToDevice));
      A_size_dirtyOnCpu = false;
      covert_search_dp_gpu_kernel3<<<b_grid, b_block>>>(*gpu_priorh_data,
        *gpu_A_size, *gpu_dpmap_size, *gpu_sizeA, pEnd, *gpu_dpmap_data);
      CUDACHECK(hipGetLastError());
      dpmap_data_dirtyOnCpu = false;
    }
  }

  l_y = NULL;
  m = emlrtCreateCharArray(2, &iv5[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m, &u[0]);
  emlrtAssign(&l_y, m);
  m_y = NULL;
  m = emlrtCreateCharArray(2, &iv6[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 10, m, &priorhfp[0]);
  emlrtAssign(&m_y, m);
  n_y = NULL;
  m = emlrtCreateString1('r');
  emlrtAssign(&n_y, m);
  nrows = emlrt_marshallIn(feval(l_y, m_y, n_y, &b_emlrtMCI),
    "<output of feval>");
  b_t = NULL;
  b_count = NULL;
  o_y = NULL;
  m = emlrtCreateCharArray(2, &iv7[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m, &b_u[0]);
  emlrtAssign(&o_y, m);
  p_y = NULL;
  m = emlrtCreateDoubleScalar(nrows);
  emlrtAssign(&p_y, m);
  q_y = NULL;
  m = emlrtCreateNumericArray(2, &iv8[0], mxDOUBLE_CLASS, mxREAL);
  pData = emlrtMxGetPr(m);
  for (i = 0; i < 2; i++) {
    pData[i] = 2400.0;
  }

  emlrtAssign(&q_y, m);
  r_y = NULL;
  m = emlrtCreateCharArray(2, &iv9[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 6, m, &precision[0]);
  emlrtAssign(&r_y, m);
  s_y = NULL;
  m = emlrtCreateNumericMatrix(1, 1, mxINT32_CLASS, mxREAL);
  *(int32_T *)emlrtMxGetData(m) = 0;
  emlrtAssign(&s_y, m);
  feval(o_y, p_y, q_y, r_y, s_y, &c_emlrtMCI, &b_t, &b_count);
  emlrt_marshallIn(emlrtAlias(b_count), "count");
  t_y = NULL;
  m = emlrtCreateDoubleScalar(1.0);
  emlrtAssign(&t_y, m);
  nrows = emlrt_marshallIn(size(emlrtAlias(b_t), t_y, &d_emlrtMCI),
    "<output of size>");
  u_y = NULL;
  m = emlrtCreateDoubleScalar(2.0);
  emlrtAssign(&u_y, m);
  ncols = emlrt_marshallIn(size(emlrtAlias(b_t), u_y, &e_emlrtMCI),
    "<output of size>");
  if (nrows * ncols == 0.0) {
    priorh_size[0] = 2400;
    priorh_size[1] = 0;
    priorh_size_dirtyOnCpu = true;
  } else {
    emlrt_marshallIn(emlrtAlias(b_t), "t", SD->f0.priorh_data, priorh_size);
    priorh_size_dirtyOnCpu = true;
    priorh_data_dirtyOnGpu = false;
    priorh_data_dirtyOnCpu = true;
  }

  emlrtDestroyArray(&b_t);
  emlrtDestroyArray(&b_count);
  v_y = NULL;
  m = emlrtCreateCharArray(2, &iv10[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m, &u[0]);
  emlrtAssign(&v_y, m);
  w_y = NULL;
  m = emlrtCreateCharArray(2, &iv11[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 8, m, &filename[0]);
  emlrtAssign(&w_y, m);
  x_y = NULL;
  m = emlrtCreateString1('r');
  emlrtAssign(&x_y, m);
  nrows = emlrt_marshallIn(feval(v_y, w_y, x_y, &b_emlrtMCI),
    "<output of feval>");
  c_t = NULL;
  c_count = NULL;
  y_y = NULL;
  m = emlrtCreateCharArray(2, &iv12[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m, &b_u[0]);
  emlrtAssign(&y_y, m);
  ab_y = NULL;
  m = emlrtCreateDoubleScalar(nrows);
  emlrtAssign(&ab_y, m);
  bb_y = NULL;
  m = emlrtCreateNumericArray(2, &iv13[0], mxDOUBLE_CLASS, mxREAL);
  pData = emlrtMxGetPr(m);
  for (i = 0; i < 2; i++) {
    pData[i] = 135.0;
  }

  emlrtAssign(&bb_y, m);
  cb_y = NULL;
  m = emlrtCreateCharArray(2, &iv14[0]);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 6, m, &precision[0]);
  emlrtAssign(&cb_y, m);
  db_y = NULL;
  m = emlrtCreateNumericMatrix(1, 1, mxINT32_CLASS, mxREAL);
  *(int32_T *)emlrtMxGetData(m) = 0;
  emlrtAssign(&db_y, m);
  feval(y_y, ab_y, bb_y, cb_y, db_y, &c_emlrtMCI, &c_t, &c_count);
  emlrt_marshallIn(emlrtAlias(c_count), "count");
  eb_y = NULL;
  m = emlrtCreateDoubleScalar(1.0);
  emlrtAssign(&eb_y, m);
  nrows = emlrt_marshallIn(size(emlrtAlias(c_t), eb_y, &d_emlrtMCI),
    "<output of size>");
  fb_y = NULL;
  m = emlrtCreateDoubleScalar(2.0);
  emlrtAssign(&fb_y, m);
  ncols = emlrt_marshallIn(size(emlrtAlias(c_t), fb_y, &e_emlrtMCI),
    "<output of size>");
  if (nrows * ncols == 0.0) {
    rtmp_size[0] = 135;
    rtmp_size[1] = 0;
    rtmp_size_dirtyOnCpu = true;
  } else {
    b_emlrt_marshallIn(emlrtAlias(c_t), "t", rtmp_data, rtmp_size);
    rtmp_size_dirtyOnCpu = true;
    rtmp_data_dirtyOnCpu = true;
  }

  emlrtDestroyArray(&c_t);
  emlrtDestroyArray(&c_count);

  //  Setup some generic parameters
  //  Fixed parameters
  rt = static_cast<int32_T>(trunc(static_cast<real_T>(rtmp_size[0]) / 2.0));

  //
  //  Create effective prior
  emxInit_int8_T(&tpaMat, 1, true);
  tpaMat_dirtyOnGpu = false;
  nrows = floor(trials / 2.0);
  nrows_dirtyOnCpu = true;
  ncols = floor(trials / 2.0);
  pEnd = tpaMat->size[0];
  tpaMat->size[0] = static_cast<int32_T>(nrows) + static_cast<int32_T>(ncols);
  emxEnsureCapacity_int8_T(tpaMat, pEnd);
  tpaMat_dirtyOnCpu = true;
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>
    ((static_cast<int32_T>(nrows) - 1) + 1L), &c_grid, &c_block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_int8_T(tpaMat, &gpu_tpaMat);
    covert_search_dp_gpu_kernel4<<<c_grid, c_block>>>(static_cast<int32_T>(nrows)
      - 1, gpu_tpaMat);
    CUDACHECK(hipGetLastError());
    tpaMat_dirtyOnCpu = false;
    tpaMat_dirtyOnGpu = true;
  }

  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>
    ((static_cast<int32_T>(ncols) - 1) + 1L), &d_grid, &d_block, 1024U, 65535U);
  if (validLaunchParams) {
    if (tpaMat_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_int8_T(tpaMat, &gpu_tpaMat);
      tpaMat_dirtyOnCpu = false;
    }

    CUDACHECK(hipMemcpy(gpu_nrows, &nrows, 8UL, hipMemcpyHostToDevice));
    nrows_dirtyOnCpu = false;
    covert_search_dp_gpu_kernel5<<<d_grid, d_block>>>(gpu_nrows,
      static_cast<int32_T>(ncols) - 1, gpu_tpaMat);
    CUDACHECK(hipGetLastError());
    tpaMat_dirtyOnGpu = true;
  }

  emxInit_real_T(&dv1, 2, true);
  sizeA_dirtyOnGpu = false;

  //  1 for target present, 0 for absent.
  pEnd = dv1->size[0] * dv1->size[1];
  dv1->size[0] = 1;
  if (tpaMat_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_int8_T(tpaMat, &gpu_tpaMat);
    tpaMat_dirtyOnGpu = false;
  }

  dv1->size[1] = tpaMat->size[0];
  emxEnsureCapacity_real_T(dv1, pEnd);
  dv1_dirtyOnCpu = true;
  if (dv1->size[1] != 0) {
    emlrtRandu(&dv1->data[0], dv1->size[1]);
  }

  emxInit_int32_T(&idx, 2, true);
  idx_dirtyOnGpu = false;
  n = dv1->size[1] + 1;
  pEnd = idx->size[0] * idx->size[1];
  idx->size[0] = 1;
  idx->size[1] = dv1->size[1];
  emxEnsureCapacity_int32_T(idx, pEnd);
  idx_dirtyOnCpu = true;
  p = dv1->size[1];
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((p - 1) + 1L),
    &e_grid, &e_block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_real_T(dv1, &gpu_dv1);
    dv1_dirtyOnCpu = false;
    gpuEmxMemcpyCpuToGpu_int32_T(idx, &gpu_idx);
    covert_search_dp_gpu_kernel6<<<e_grid, e_block>>>(p, gpu_idx);
    CUDACHECK(hipGetLastError());
    idx_dirtyOnCpu = false;
    idx_dirtyOnGpu = true;
  }

  if (dv1->size[1] != 0) {
    emxInit_int32_T(&iwork, 1, true);
    pEnd = iwork->size[0];
    iwork->size[0] = dv1->size[1];
    emxEnsureCapacity_int32_T(iwork, pEnd);
    iwork_dirtyOnCpu = true;
    pEnd = dv1->size[1] - 1;
    validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((pEnd - 1) / 2
      + 1L), &f_grid, &f_block, 1024U, 65535U);
    if (validLaunchParams) {
      if (dv1_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(dv1, &gpu_dv1);
        dv1_dirtyOnCpu = false;
      }

      if (idx_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_int32_T(idx, &gpu_idx);
        idx_dirtyOnCpu = false;
      }

      covert_search_dp_gpu_kernel7<<<f_grid, f_block>>>(gpu_dv1, pEnd, gpu_idx);
      CUDACHECK(hipGetLastError());
      idx_dirtyOnGpu = true;
    }

    if ((dv1->size[1] & 1) != 0) {
      p = dv1->size[1];
      if (dv1_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(dv1, &gpu_dv1);
      }

      if (idx_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_int32_T(idx, &gpu_idx);
        idx_dirtyOnCpu = false;
      }

      covert_search_dp_gpu_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(p,
        gpu_idx);
      CUDACHECK(hipGetLastError());
      idx_dirtyOnGpu = true;
    }

    i = 2;
    while (i < n - 1) {
      i2 = i << 1;
      j = 1;
      for (pEnd = i + 1; pEnd < n; pEnd = qEnd + i) {
        p = j;
        q = pEnd - 1;
        qEnd = j + i2;
        if (qEnd > n) {
          qEnd = n;
        }

        k = 0;
        kEnd = qEnd - j;
        while (k + 1 <= kEnd) {
          if (idx_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_int32_T(idx, &gpu_idx);
            idx_dirtyOnGpu = false;
          }

          if ((dv1->data[idx->data[p - 1] - 1] <= dv1->data[idx->data[q] - 1]) ||
              rtIsNaN(dv1->data[idx->data[q] - 1])) {
            iwork->data[k] = idx->data[p - 1];
            iwork_dirtyOnCpu = true;
            p++;
            if (p == pEnd) {
              while (q + 1 < qEnd) {
                k++;
                iwork->data[k] = idx->data[q];
                q++;
              }
            }
          } else {
            iwork->data[k] = idx->data[q];
            iwork_dirtyOnCpu = true;
            q++;
            if (q + 1 == qEnd) {
              while (p < pEnd) {
                k++;
                iwork->data[k] = idx->data[p - 1];
                p++;
              }
            }
          }

          k++;
        }

        validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((kEnd - 1)
          + 1L), &g_grid, &g_block, 1024U, 65535U);
        if (validLaunchParams) {
          if (idx_dirtyOnCpu) {
            gpuEmxMemcpyCpuToGpu_int32_T(idx, &gpu_idx);
            idx_dirtyOnCpu = false;
          }

          if (iwork_dirtyOnCpu) {
            gpuEmxMemcpyCpuToGpu_int32_T(iwork, &gpu_iwork);
            iwork_dirtyOnCpu = false;
          }

          covert_search_dp_gpu_kernel9<<<g_grid, g_block>>>(gpu_iwork, j, kEnd,
            gpu_idx);
          CUDACHECK(hipGetLastError());
          idx_dirtyOnGpu = true;
        }

        j = qEnd;
      }

      i = i2;
    }

    emxFree_int32_T(&iwork);
    gpuEmxFree_int32_T(&gpu_iwork);
  }

  for (i = 0; i < 2; i++) {
    A_size[i] = dv1->size[i];
    A_size_dirtyOnCpu = true;
  }

  pEnd = dv1->size[0] * dv1->size[1];
  dv1->size[0] = 1;
  dv1->size[1] = A_size[1];
  emxEnsureCapacity_real_T(dv1, pEnd);
  dv1_dirtyOnCpu = true;
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((A_size[0] *
    A_size[1] - 1) + 1L), &h_grid, &h_block, 1024U, 65535U);
  if (validLaunchParams) {
    if (A_size_dirtyOnCpu) {
      CUDACHECK(hipMemcpy(gpu_A_size, &A_size[0], 8UL, hipMemcpyHostToDevice));
    }

    gpuEmxMemcpyCpuToGpu_real_T(dv1, &gpu_dv1);
    if (idx_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_int32_T(idx, &gpu_idx);
    }

    covert_search_dp_gpu_kernel10<<<h_grid, h_block>>>(gpu_idx, *gpu_A_size,
      gpu_dv1);
    CUDACHECK(hipGetLastError());
    dv1_dirtyOnCpu = false;
    sizeA_dirtyOnGpu = true;
  }

  emxFree_int32_T(&idx);
  gpuEmxFree_int32_T(&gpu_idx);
  emxInit_int8_T(&b_tpaMat, 1, true);
  idx_dirtyOnGpu = false;
  pEnd = b_tpaMat->size[0];
  if (sizeA_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(dv1, &gpu_dv1);
  }

  b_tpaMat->size[0] = dv1->size[1];
  emxEnsureCapacity_int8_T(b_tpaMat, pEnd);
  A_size_dirtyOnCpu = true;
  p = dv1->size[1];
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((p - 1) + 1L),
    &i_grid, &i_block, 1024U, 65535U);
  if (validLaunchParams) {
    if (tpaMat_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_int8_T(tpaMat, &gpu_tpaMat);
    }

    if (dv1_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_real_T(dv1, &gpu_dv1);
    }

    gpuEmxMemcpyCpuToGpu_int8_T(b_tpaMat, &b_gpu_tpaMat);
    covert_search_dp_gpu_kernel11<<<i_grid, i_block>>>(gpu_tpaMat, gpu_dv1, p,
      b_gpu_tpaMat);
    CUDACHECK(hipGetLastError());
    A_size_dirtyOnCpu = false;
    idx_dirtyOnGpu = true;
  }

  emxFree_real_T(&dv1);
  gpuEmxFree_real_T(&gpu_dv1);
  pEnd = tpaMat->size[0];
  if (idx_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_int8_T(b_tpaMat, &b_gpu_tpaMat);
  }

  tpaMat->size[0] = b_tpaMat->size[0];
  emxEnsureCapacity_int8_T(tpaMat, pEnd);
  p = b_tpaMat->size[0];
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((p - 1) + 1L),
    &j_grid, &j_block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_int8_T(tpaMat, &gpu_tpaMat);
    if (A_size_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_int8_T(b_tpaMat, &b_gpu_tpaMat);
    }

    covert_search_dp_gpu_kernel12<<<j_grid, j_block>>>(b_gpu_tpaMat, p,
      gpu_tpaMat);
    CUDACHECK(hipGetLastError());
    tpaMat_dirtyOnGpu = true;
  }

  emxFree_int8_T(&b_tpaMat);
  gpuEmxFree_int8_T(&b_gpu_tpaMat);

  //  permute isn't necessary becuase this search with no memory. we permute anyway. 
  //  integer pixel locations
  if (priorh_size[0] * priorh_size[1] != 0) {
    pEnd = priorh_size[0] * priorh_size[1];
    if (priorh_data_dirtyOnCpu) {
      CUDACHECK(hipMemcpy(gpu_priorh_data, &SD->f0.priorh_data[0], 46080000UL,
                           hipMemcpyHostToDevice));
    }

    if (nrows_dirtyOnCpu) {
      CUDACHECK(hipMemcpy(gpu_nrows, &nrows, 8UL, hipMemcpyHostToDevice));
    }

    covert_search_dp_gpu_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_priorh_data, gpu_nrows);
    CUDACHECK(hipGetLastError());
    A_size_dirtyOnCpu = true;
    validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((pEnd - 2) +
      1L), &k_grid, &k_block, 1024U, 65535U);
    if (validLaunchParams) {
      CUDACHECK(hipMemcpy(gpu_priorh_size, &priorh_size[0], 8UL,
                           hipMemcpyHostToDevice));
      priorh_size_dirtyOnCpu = false;
      covert_search_dp_gpu_kernel14<<<k_grid, k_block>>>(*gpu_priorh_data, pEnd,
        gpu_nrows);
      CUDACHECK(hipGetLastError());
    }

    edges_size[1] = priorh_size[0] * priorh_size[1] + 1;
    SD->f0.edges_data[0] = 0.0;
    SD->f0.edges_data[priorh_size[0] * priorh_size[1]] = 1.0;
    pEnd = priorh_size[0] * priorh_size[1];
    for (j = 0; j <= pEnd - 2; j++) {
      if (priorh_data_dirtyOnGpu) {
        CUDACHECK(hipMemcpy(&SD->f0.priorh_data[0], gpu_priorh_data, 46080000UL,
                             hipMemcpyDeviceToHost));
        priorh_data_dirtyOnGpu = false;
      }

      if (A_size_dirtyOnCpu) {
        CUDACHECK(hipMemcpy(&nrows, gpu_nrows, 8UL, hipMemcpyDeviceToHost));
        A_size_dirtyOnCpu = false;
      }

      SD->f0.edges_data[j + 1] = fmin(SD->f0.edges_data[j] + SD->
        f0.priorh_data[j] / nrows, 1.0);
    }
  } else {
    edges_size[1] = 1;
    SD->f0.edges_data[0] = 0.0;
  }

  emxInit_real_T(&b_idx, 1, true);
  idx_dirtyOnGpu = false;
  pEnd = b_idx->size[0];
  b_idx->size[0] = static_cast<int32_T>(trials);
  emxEnsureCapacity_real_T(b_idx, pEnd);
  idx_dirtyOnCpu = true;
  if (static_cast<int32_T>(trials) > 0) {
    if (priorh_size[0] * priorh_size[1] == 0) {
      pEnd = b_idx->size[0];
      b_idx->size[0] = static_cast<int32_T>(trials);
      emxEnsureCapacity_real_T(b_idx, pEnd);
      emlrtRandu(&b_idx->data[0], b_idx->size[0]);
      pEnd = b_idx->size[0];
      validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((pEnd - 1) +
        1L), &l_grid, &l_block, 1024U, 65535U);
      if (validLaunchParams) {
        if (priorh_size_dirtyOnCpu) {
          CUDACHECK(hipMemcpy(gpu_priorh_size, &priorh_size[0], 8UL,
                               hipMemcpyHostToDevice));
          priorh_size_dirtyOnCpu = false;
        }

        gpuEmxMemcpyCpuToGpu_real_T(b_idx, &b_gpu_idx);
        covert_search_dp_gpu_kernel16<<<l_grid, l_block>>>(*gpu_priorh_size,
          pEnd, b_gpu_idx);
        CUDACHECK(hipGetLastError());
        idx_dirtyOnCpu = false;
        idx_dirtyOnGpu = true;
      }
    } else {
      emxInit_real_T(&r, 1, true);
      pEnd = r->size[0];
      r->size[0] = static_cast<int32_T>(trials);
      emxEnsureCapacity_real_T(r, pEnd);
      emlrtRandu(&r->data[0], r->size[0]);
      dv2[0] = static_cast<uint32_T>(r->size[0]);
      pEnd = b_idx->size[0];
      b_idx->size[0] = static_cast<int32_T>(dv2[0]);
      emxEnsureCapacity_real_T(b_idx, pEnd);
      validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((
        static_cast<int32_T>(dv2[0]) - 1) + 1L), &l_grid, &l_block, 1024U,
        65535U);
      if (validLaunchParams) {
        CUDACHECK(hipMemcpy(gpu_dv2, &dv2[0], 8UL, hipMemcpyHostToDevice));
        gpuEmxMemcpyCpuToGpu_real_T(b_idx, &b_gpu_idx);
        covert_search_dp_gpu_kernel15<<<l_grid, l_block>>>(*gpu_dv2, b_gpu_idx);
        CUDACHECK(hipGetLastError());
        idx_dirtyOnCpu = false;
        idx_dirtyOnGpu = true;
      }

      pEnd = 0;
      p = r->size[0];
      for (k = 0; k < p; k++) {
        i2 = 0;
        if (!rtIsNaN(r->data[pEnd])) {
          if ((r->data[pEnd] >= SD->f0.edges_data[0]) && (r->data[pEnd] <
               SD->f0.edges_data[edges_size[1] - 1])) {
            q = edges_size[1];
            i2 = 1;
            qEnd = 2;
            while (q > qEnd) {
              kEnd = (i2 >> 1) + (q >> 1);
              if (((i2 & 1) == 1) && ((q & 1) == 1)) {
                kEnd++;
              }

              if (r->data[pEnd] >= SD->f0.edges_data[kEnd - 1]) {
                i2 = kEnd;
                qEnd = kEnd + 1;
              } else {
                q = kEnd;
              }
            }
          }

          if (r->data[pEnd] == SD->f0.edges_data[edges_size[1] - 1]) {
            i2 = edges_size[1];
          }
        }

        if (idx_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(b_idx, &b_gpu_idx);
          idx_dirtyOnGpu = false;
        }

        b_idx->data[pEnd] = i2;
        idx_dirtyOnCpu = true;
        pEnd++;
      }

      emxFree_real_T(&r);
    }
  }

  if (priorh_size_dirtyOnCpu) {
    CUDACHECK(hipMemcpy(gpu_priorh_size, &priorh_size[0], 8UL,
                         hipMemcpyHostToDevice));
  }

  covert_search_dp_gpu_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_priorh_size, *gpu_sizeA);
  CUDACHECK(hipGetLastError());
  emxInit_int32_T(&v1, 1, true);
  nrows_dirtyOnCpu = false;
  pEnd = v1->size[0];
  if (idx_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(b_idx, &b_gpu_idx);
  }

  v1->size[0] = b_idx->size[0];
  emxEnsureCapacity_int32_T(v1, pEnd);
  A_size_dirtyOnCpu = true;
  p = b_idx->size[0];
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((p - 1) + 1L),
    &m_grid, &m_block, 1024U, 65535U);
  if (validLaunchParams) {
    if (idx_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_real_T(b_idx, &b_gpu_idx);
    }

    gpuEmxMemcpyCpuToGpu_int32_T(v1, &gpu_v1);
    covert_search_dp_gpu_kernel18<<<m_grid, m_block>>>(b_gpu_idx, p, gpu_v1);
    CUDACHECK(hipGetLastError());
    A_size_dirtyOnCpu = false;
    nrows_dirtyOnCpu = true;
  }

  emxFree_real_T(&b_idx);
  gpuEmxFree_real_T(&b_gpu_idx);
  emxInit_int32_T(&vk, 1, true);
  idx_dirtyOnGpu = false;
  CUDACHECK(hipMemcpy(&sizeA[0], gpu_sizeA, 4UL, hipMemcpyDeviceToHost));
  pEnd = vk->size[0];
  if (nrows_dirtyOnCpu) {
    gpuEmxMemcpyGpuToCpu_int32_T(v1, &gpu_v1);
    nrows_dirtyOnCpu = false;
  }

  vk->size[0] = v1->size[0];
  emxEnsureCapacity_int32_T(vk, pEnd);
  iwork_dirtyOnCpu = true;
  p = v1->size[0];
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((p - 1) + 1L),
    &n_grid, &n_block, 1024U, 65535U);
  if (validLaunchParams) {
    if (A_size_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_int32_T(v1, &gpu_v1);
    }

    gpuEmxMemcpyCpuToGpu_int32_T(vk, &gpu_vk);
    covert_search_dp_gpu_kernel19<<<n_grid, n_block>>>(static_cast<int32_T>
      (sizeA[0]), gpu_v1, p, gpu_vk);
    CUDACHECK(hipGetLastError());
    iwork_dirtyOnCpu = false;
    idx_dirtyOnGpu = true;
  }

  pEnd = v1->size[0];
  emxEnsureCapacity_int32_T(v1, pEnd);
  A_size_dirtyOnCpu = true;
  p = v1->size[0];
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((p - 1) + 1L),
    &o_grid, &o_block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_int32_T(v1, &gpu_v1);
    if (iwork_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_int32_T(vk, &gpu_vk);
      iwork_dirtyOnCpu = false;
    }

    covert_search_dp_gpu_kernel20<<<o_grid, o_block>>>(static_cast<int32_T>
      (sizeA[0]), gpu_vk, p, gpu_v1);
    CUDACHECK(hipGetLastError());
    A_size_dirtyOnCpu = false;
    nrows_dirtyOnCpu = true;
  }

  emxInit_int32_T(&pre_yt, 1, true);
  sizeA_dirtyOnGpu = false;
  pEnd = pre_yt->size[0];
  if (nrows_dirtyOnCpu) {
    gpuEmxMemcpyGpuToCpu_int32_T(v1, &gpu_v1);
  }

  pre_yt->size[0] = v1->size[0];
  emxEnsureCapacity_int32_T(pre_yt, pEnd);
  p = v1->size[0];
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((p - 1) + 1L),
    &p_grid, &p_block, 1024U, 65535U);
  if (validLaunchParams) {
    if (A_size_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_int32_T(v1, &gpu_v1);
    }

    gpuEmxMemcpyCpuToGpu_int32_T(pre_yt, &gpu_pre_yt);
    covert_search_dp_gpu_kernel21<<<p_grid, p_block>>>(gpu_v1, p, gpu_pre_yt);
    CUDACHECK(hipGetLastError());
    sizeA_dirtyOnGpu = true;
  }

  emxFree_int32_T(&v1);
  gpuEmxFree_int32_T(&gpu_v1);
  emxInit_int32_T(&pre_xt, 1, true);
  A_size_dirtyOnCpu = false;
  pEnd = pre_xt->size[0];
  if (idx_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_int32_T(vk, &gpu_vk);
  }

  pre_xt->size[0] = vk->size[0];
  emxEnsureCapacity_int32_T(pre_xt, pEnd);
  p = vk->size[0];
  validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((p - 1) + 1L),
    &q_grid, &q_block, 1024U, 65535U);
  if (validLaunchParams) {
    if (iwork_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_int32_T(vk, &gpu_vk);
    }

    gpuEmxMemcpyCpuToGpu_int32_T(pre_xt, &gpu_pre_xt);
    covert_search_dp_gpu_kernel22<<<q_grid, q_block>>>(gpu_vk, p, gpu_pre_xt);
    CUDACHECK(hipGetLastError());
    A_size_dirtyOnCpu = true;
  }

  emxFree_int32_T(&vk);
  gpuEmxFree_int32_T(&gpu_vk);

  //  Run simulation loop
  pEnd = static_cast<int32_T>(trials);
  for (k = 0; k < pEnd; k++) {
    //  background number
    emlrtRandn(&SD->f0.re[0], 360000);

    //  human experiments have 30 independent pix per degree. upsample via nearest neighbor to run simulations in 120 pix per degree. 
    //  sneaky way to increase the matrix with blocks -- Kronecker product
    if (tpaMat_dirtyOnGpu) {
      gpuEmxMemcpyGpuToCpu_int8_T(tpaMat, &gpu_tpaMat);
      tpaMat_dirtyOnGpu = false;
    }

    if (tpaMat->data[k] != 0) {
      //  if target present add target responses
      if (sizeA_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_int32_T(pre_yt, &gpu_pre_yt);
        sizeA_dirtyOnGpu = false;
      }

      nrows = static_cast<real_T>(pre_yt->data[k]) - static_cast<real_T>(rt);
      ncols = static_cast<real_T>(pre_yt->data[k]) + static_cast<real_T>(rt);
      if (nrows > ncols) {
        p = 0;
        q = -1;
      } else {
        p = static_cast<int32_T>(nrows) - 1;
        q = static_cast<int32_T>(ncols) - 1;
      }

      if (A_size_dirtyOnCpu) {
        gpuEmxMemcpyGpuToCpu_int32_T(pre_xt, &gpu_pre_xt);
        A_size_dirtyOnCpu = false;
      }

      nrows = static_cast<real_T>(pre_xt->data[k]) - static_cast<real_T>(rt);
      ncols = static_cast<real_T>(pre_xt->data[k]) + static_cast<real_T>(rt);
      if (nrows > ncols) {
        i2 = 0;
        qEnd = -1;
      } else {
        i2 = static_cast<int32_T>(nrows) - 1;
        qEnd = static_cast<int32_T>(ncols) - 1;
      }

      x_size[0] = (q - p) + 1;
      x_size[1] = (qEnd - i2) + 1;
      validLaunchParams = mwGetLaunchParameters(static_cast<real_T>(((q - p) +
        1L) * ((qEnd - i2) + 1L)), &r_grid, &r_block, 1024U, 65535U);
      if (validLaunchParams) {
        if (dpmap_data_dirtyOnCpu) {
          CUDACHECK(hipMemcpy(gpu_dpmap_data, &SD->f0.dpmap_data[0], 46080000UL,
                               hipMemcpyHostToDevice));
          dpmap_data_dirtyOnCpu = false;
        }

        if (dpmap_size_dirtyOnCpu) {
          CUDACHECK(hipMemcpy(gpu_dpmap_size, &dpmap_size[0], 8UL,
                               hipMemcpyHostToDevice));
          dpmap_size_dirtyOnCpu = false;
        }

        if (rtmp_data_dirtyOnCpu) {
          CUDACHECK(hipMemcpy(gpu_rtmp_data, &rtmp_data[0], 145800UL,
                               hipMemcpyHostToDevice));
          rtmp_data_dirtyOnCpu = false;
        }

        if (rtmp_size_dirtyOnCpu) {
          CUDACHECK(hipMemcpy(gpu_rtmp_size, &rtmp_size[0], 8UL,
                               hipMemcpyHostToDevice));
          rtmp_size_dirtyOnCpu = false;
        }

        CUDACHECK(hipMemcpy(gpu_x_size, &x_size[0], 8UL, hipMemcpyHostToDevice));
      }

      //  create the local target response
    }

    //  compute normalized responses
    //  compute log likelihood ratio of target present,
    //  location of max normalized response, and max normalized response
    // log(sum(eslr(:), 'omitnan'));
  }

  emxFree_int32_T(&pre_xt);
  gpuEmxFree_int32_T(&gpu_pre_xt);
  emxFree_int32_T(&pre_yt);
  gpuEmxFree_int32_T(&gpu_pre_yt);
  emxFree_int8_T(&tpaMat);
  gpuEmxFree_int8_T(&gpu_tpaMat);

  //  Structured Data Storage
  // results = 1;
  // results = table(trial, stimY, stimX, tPresent, ymax, xmax, llr, slpmx,...
  //     'VariableNames', {'trial', 'stimY', 'stimX', 'tPresent', 'ymax', 'xmax', 'llr', 'slpmx'}); 
  coder::toc();
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  CUDACHECK(hipFree(gpu_nrows));
  CUDACHECK(hipFree(*gpu_dpmap_size));
  CUDACHECK(hipFree(*gpu_A_size));
  CUDACHECK(hipFree(*gpu_dpmap_data));
  CUDACHECK(hipFree(*gpu_sizeA));
  CUDACHECK(hipFree(*gpu_priorh_data));
  CUDACHECK(hipFree(*gpu_priorh_size));
  CUDACHECK(hipFree(*gpu_rtmp_size));
  CUDACHECK(hipFree(*gpu_dv2));
  CUDACHECK(hipFree(*gpu_x_size));
  CUDACHECK(hipFree(*gpu_rtmp_data));

#undef CUDACHECK

}

// End of code generation (covert_search_dp_gpu.cu)
