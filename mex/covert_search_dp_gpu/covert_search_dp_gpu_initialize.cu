//
//  Academic License - for use in teaching, academic research, and meeting
//  course requirements at degree granting institutions only.  Not for
//  government, commercial, or other organizational use.
//
//  covert_search_dp_gpu_initialize.cu
//
//  Code generation for function 'covert_search_dp_gpu_initialize'
//


// Include files
#include "covert_search_dp_gpu_initialize.h"
#include "_coder_covert_search_dp_gpu_mex.h"
#include "covert_search_dp_gpu_data.h"
#include "rt_nonfinite.h"
#include "timeKeeper.h"

// Function Declarations
static void covert_search_dp_gpu_once();

// Function Definitions
static void covert_search_dp_gpu_once()
{
  mex_InitInfAndNan();
  savedTime_not_empty_init();
}

void covert_search_dp_gpu_initialize()
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "image_toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "statistics_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    covert_search_dp_gpu_once();
  }

  hipGetLastError();
}

// End of code generation (covert_search_dp_gpu_initialize.cu)
